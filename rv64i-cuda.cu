
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stddef.h>
#include <assert.h>
#include <stdbool.h>
#include <string.h>

typedef int8_t int8;
typedef uint8_t uint8;
typedef int16_t int16;
typedef uint16_t uint16;
typedef int32_t int32;
typedef uint32_t uint32;
typedef int64_t int64;
typedef uint64_t uint64;

#define UCYCLE 0x320
#define UHALT 0x328
#define UPC 0x330
#define UX0 0x340


struct UarchState {
    uint64 access_paddr[16];
    uint64 access_val[16];
    uint8 access_readWriteEnd[16];
    uint8 access_pointer;
    uint8 trap;
};

typedef struct UarchState UarchState;

enum UArchStepStatus {
    Success,       // one micro instruction was executed successfully
    CycleOverflow, // already at fixed point: uarch cycle has reached its maximum value
    UArchHalted    // already at fixed point: microarchitecture is halted
};


__device__ inline uint64 readWord(UarchState *a, uint64 paddr) {
    if (a->access_pointer > 16) {
       a->trap = 18;
       return 0;
    }
    if (a->access_readWriteEnd[a->access_pointer] == 0 && a->access_paddr[a->access_pointer] == paddr) {
       return (a->access_val[a->access_pointer++]);
    } else {
       a->access_pointer++;
       a->trap = 19;
       
       return 0;
    }
}

__device__ inline void writeWord(UarchState *a, uint64 paddr, uint64 val) {
    if (a->access_pointer > 16) {
       a->trap = 20;
       return;
    }
    if (a->access_readWriteEnd[a->access_pointer] == 1 && a->access_paddr[a->access_pointer] == paddr && a->access_val[a->access_pointer] == val) {
      a->access_pointer++;
    } else {
      a->access_pointer++;
      a->trap = 21;
    }
}

__device__ inline uint64 readCycle(UarchState *a) {
    return readWord(a, UCYCLE);
}

__device__ inline void writeCycle(UarchState *a, uint64 val) {
    writeWord(a, UCYCLE, val);
}

__device__ inline bool readHaltFlag(UarchState *a) {
    return readWord(a, UHALT) != 0;
}

__device__ inline void setHaltFlag(UarchState *a) {
    writeWord(a, UHALT, 1);
}

__device__ inline uint64 readPc(UarchState *a) {
    return readWord(a, UPC);
}

__device__ inline void writePc(UarchState *a, uint64 val) {
    writeWord(a, UPC, val);
}

__device__ inline uint64 readX(UarchState *a, uint8 reg) {
    return readWord(a, UX0 + (reg << 3));
}

__device__ inline void writeX(UarchState *a, uint8 reg, uint64 val) {
    writeWord(a, UX0 + (reg << 3), val);
}


__device__ void require(UarchState *a, bool condition, const char *message) {
    if (!condition) {
      a->trap = 1;
    }
    //assert((condition) && (message));
}

__device__ void dumpInsn(UarchState *a, uint64 pc, uint32 insn, const char *name) {
}

__device__ inline int32 uint64ToInt32(uint64 v) {
    return (int32)(v);
}

__device__ inline uint64 uint64AddInt32(uint64 v, int32 w) {
    return v + w;
}

__device__ inline uint64 uint64SubUInt64(uint64 v, uint64 w) {
    return v - w;
}

__device__ inline uint64 uint64AddUInt64(uint64 v, uint64 w) {
    return v + w;
}

__device__ inline uint64 uint64ShiftRight(uint64 v, uint32 count) {
    return v >> (count & 0x3f);
}

__device__ inline uint64 uint64ShiftLeft(uint64 v, uint32 count) {
    return v << (count & 0x3f);
}

__device__ inline int64 int64ShiftRight(int64 v, uint32 count) {
    return v >> (count & 0x3f);
}

__device__ inline int64 int64AddInt64(int64 v, int64 w) {
    //int64 res = 0;
    return v+w;
}

__device__ inline uint32 uint32ShiftRight(uint32 v, uint32 count) {
    return v >> (count & 0x1f);
}

__device__ inline uint32 uint32ShiftLeft(uint32 v, uint32 count) {
    return v << (count & 0x1f);
}

__device__ inline uint64 int32ToUInt64(int32 v) {
    return v;
}

__device__ inline int32 int32ShiftRight(int32 v, uint32 count) {
    return v >> (count & 0x1f);
}

__device__ inline int32 int32AddInt32(int32 v, int32 w) {
//    int32 res = 0;
//    __builtin_add_overflow(v, w, &res);
    return v+w;
}

__device__ inline int32 int32SubInt32(int32 v, int32 w) {
    return v-w;
}

__device__ inline uint64 int16ToUInt64(int16 v) {
    return v;
}

__device__ inline uint64 int8ToUInt64(int8 v) {
    return v;
}

// Memory read/write access

__device__ inline uint64 readUInt64(UarchState *a, uint64 paddr) {
    require(a, (paddr & 7) == 0, "misaligned readUInt64 address");
    return readWord(a, paddr);
}


__device__ inline uint32 readUInt32(UarchState *a, uint64 paddr) {
    require(a, (paddr & 3) == 0, "misaligned readUInt32 address");
    uint64 palign = paddr & ~(uint64)(7);
    uint32 bitoffset = uint32ShiftLeft((uint32)(paddr) & (uint32)(7), 3);
    uint64 val64 = readUInt64(a, palign);
    return (uint32)(uint64ShiftRight(val64, bitoffset));
}


__device__ inline uint16 readUInt16(UarchState *a, uint64 paddr) {
    require(a, (paddr & 1) == 0, "misaligned readUInt16 address");
    uint64 palign = paddr & ~(uint64)(7);
    uint32 bitoffset = uint32ShiftLeft((uint32)(paddr) & (uint32)(7), 3);
    uint64 val64 = readUInt64(a, palign);
    return (uint16)(uint64ShiftRight(val64, bitoffset));
}


__device__ inline uint8 readUInt8(UarchState *a, uint64 paddr) {
    uint64 palign = paddr & ~(uint64)(7);
    uint32 bitoffset = uint32ShiftLeft((uint32)(paddr) & (uint32)(7), 3);
    uint64 val64 = readUInt64(a, palign);
    return (uint8)(uint64ShiftRight(val64, bitoffset));
}


__device__ inline void writeUInt64(UarchState *a, uint64 paddr, uint64 val) {
    require(a, (paddr & 7) == 0, "misaligned writeUInt64 address");
    writeWord(a, paddr, val);
}

/// \brief Copies bits from a uint64 word, starting at bit 0, to another uint64 word at the specified bit offset.
/// \param from Source of bits to copy, starting at offset 0.
/// \param count Number of bits to copy.
/// \param to Destination of copy.
/// \param offset Bit offset in destination to copy bits to.
/// \return The uint64 word containing the copy result.
__device__ inline uint64 copyBits(UarchState *a, uint32 from, uint32 count, uint64 to, uint32 offset) {
    require(a, offset + count <= 64, "copyBits count exceeds limit of 64");
    uint64 eraseMask = uint64ShiftLeft(1, count) - 1;
    eraseMask = ~uint64ShiftLeft(eraseMask, offset);
    return uint64ShiftLeft(from, offset) | (to & eraseMask);
}


__device__ inline void writeUInt32(UarchState *a, uint64 paddr, uint32 val) {
    require(a, (paddr & 3) == 0, "misaligned writeUInt32 address");
    uint64 palign = paddr & ~(uint64)(7);

    uint32 bitoffset = uint32ShiftLeft((uint32)(paddr) & (uint32)(7), 3);
    uint64 oldval64 = readUInt64(a, palign);
    uint64 newval64 = copyBits(a, val, 32, oldval64, bitoffset);
    writeUInt64(a, palign, newval64);
}


__device__ inline void writeUInt16(UarchState *a, uint64 paddr, uint16 val) {
    require(a, (paddr & 1) == 0, "misaligned writeUInt16 address");
    uint64 palign = paddr & ~(uint64)(7);
    uint32 bitoffset = uint32ShiftLeft((uint32)(paddr) & (uint32)(7), 3);
    uint64 oldval64 = readUInt64(a, palign);
    uint64 newval64 = copyBits(a, val, 16, oldval64, bitoffset);
    writeUInt64(a, palign, newval64);
}


__device__ inline void writeUInt8(UarchState *a, uint64 paddr, uint8 val) {
    uint64 palign = paddr & ~(uint64)(7);
    uint32 bitoffset = uint32ShiftLeft((uint32)(paddr) & (uint32)(7), 3);
    uint64 oldval64 = readUInt64(a, palign);
    uint64 newval64 = copyBits(a, val, 8, oldval64, bitoffset);
    writeUInt64(a, palign, newval64);
}

// Instruction operand decoders

__device__ inline uint8 operandRd(uint32 insn) {
    return (uint8)(uint32ShiftRight(uint32ShiftLeft(insn, 20), 27));
}

__device__ inline uint8 operandRs1(uint32 insn) {
    return (uint8)(uint32ShiftRight(uint32ShiftLeft(insn, 12), 27));
}

__device__ inline uint8 operandRs2(uint32 insn) {
    return (uint8)(uint32ShiftRight(uint32ShiftLeft(insn, 7), 27));
}

__device__ inline int32 operandImm12(uint32 insn) {
    return int32ShiftRight((int32)(insn), 20);
}

__device__ inline int32 operandImm20(uint32 insn) {
    return (int32)(uint32ShiftLeft(uint32ShiftRight(insn, 12), 12));
}

__device__ inline int32 operandJimm20(uint32 insn) {
    int32 a = (int32)(uint32ShiftLeft((uint32)(int32ShiftRight((int32)(insn), 31)), 20));
    uint32 b = uint32ShiftLeft(uint32ShiftRight(uint32ShiftLeft(insn, 1), 22), 1);
    uint32 c = uint32ShiftLeft(uint32ShiftRight(uint32ShiftLeft(insn, 11), 31), 11);
    uint32 d = uint32ShiftLeft(uint32ShiftRight(uint32ShiftLeft(insn, 12), 24), 12);
    return (int32)((uint32)(a) | b | c | d);
}

__device__ inline int32 operandShamt5(uint32 insn) {
    return (int32)(uint32ShiftRight(uint32ShiftLeft(insn, 7), 27));
}

__device__ inline int32 operandShamt6(uint32 insn) {
    return (int32)(uint32ShiftRight(uint32ShiftLeft(insn, 6), 26));
}

__device__ inline int32 operandSbimm12(uint32 insn) {
    int32 a = (int32)(uint32ShiftLeft((uint32)(int32ShiftRight((int32)(insn), 31)), 12));
    uint32 b = uint32ShiftLeft(uint32ShiftRight(uint32ShiftLeft(insn, 1), 26), 5);
    uint32 c = uint32ShiftLeft(uint32ShiftRight(uint32ShiftLeft(insn, 20), 28), 1);
    uint32 d = uint32ShiftLeft(uint32ShiftRight(uint32ShiftLeft(insn, 24), 31), 11);
    return (int32)((uint32)(a) | b | c | d);
}

__device__ inline int32 operandSimm12(uint32 insn) {
    return (int32)(
        uint32ShiftLeft((uint32)(int32ShiftRight((int32)(insn), 25)), 5) | uint32ShiftRight(uint32ShiftLeft(insn, 20), 27));
}

// Execute instruction


__device__ inline void advancePc(UarchState *a, uint64 pc) {
    uint64 newPc = uint64AddUInt64(pc, 4);
    return writePc(a, newPc);
}


__device__ inline void branch(UarchState *a, uint64 pc) {
    return writePc(a, pc);
}


__device__ inline void executeLUI(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "lui");
    uint8 rd = operandRd(insn);
    int32 imm = operandImm20(insn);
    if (rd != 0) {
        writeX(a, rd, int32ToUInt64(imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeAUIPC(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "auipc");
    int32 imm = operandImm20(insn);
    uint8 rd = operandRd(insn);
    if (rd != 0) {
        writeX(a, rd, uint64AddInt32(pc, imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeJAL(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "jal");
    int32 imm = operandJimm20(insn);
    uint8 rd = operandRd(insn);
    if (rd != 0) {
        writeX(a, rd, uint64AddUInt64(pc, 4));
    }
    return branch(a, uint64AddInt32(pc, imm));
}


__device__ inline void executeJALR(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "jalr");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint64 rs1val = readX(a, rs1);
    if (rd != 0) {
        writeX(a, rd, uint64AddUInt64(pc, 4));
    }
    return branch(a, uint64AddInt32(rs1val, imm) & (~(uint64)(1)));
}


__device__ inline void executeBEQ(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "beq");
    int32 imm = operandSbimm12(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    uint64 rs1val = readX(a, rs1);
    uint64 rs2val = readX(a, rs2);
    if (rs1val == rs2val) {
        return branch(a, uint64AddInt32(pc, imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeBNE(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "bne");
    int32 imm = operandSbimm12(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    uint64 rs1val = readX(a, rs1);
    uint64 rs2val = readX(a, rs2);
    if (rs1val != rs2val) {
        return branch(a, uint64AddInt32(pc, imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeBLT(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "blt");
    int32 imm = operandSbimm12(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    int64 rs1val = (int64)(readX(a, rs1));
    int64 rs2val = (int64)(readX(a, rs2));
    if (rs1val < rs2val) {
        return branch(a, uint64AddInt32(pc, imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeBGE(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "bge");
    int32 imm = operandSbimm12(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    int64 rs1val = (int64)(readX(a, rs1));
    int64 rs2val = (int64)(readX(a, rs2));
    if (rs1val >= rs2val) {
        return branch(a, uint64AddInt32(pc, imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeBLTU(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "bltu");
    int32 imm = operandSbimm12(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    uint64 rs1val = readX(a, rs1);
    uint64 rs2val = readX(a, rs2);
    if (rs1val < rs2val) {
        return branch(a, uint64AddInt32(pc, imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeBGEU(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "bgeu");
    int32 imm = operandSbimm12(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    uint64 rs1val = readX(a, rs1);
    uint64 rs2val = readX(a, rs2);
    if (rs1val >= rs2val) {
        return branch(a, uint64AddInt32(pc, imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeLB(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "lb");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint64 rs1val = readX(a, rs1);
    int8 i8 = (int8)(readUInt8(a, uint64AddInt32(rs1val, imm)));
    if (rd != 0) {
        writeX(a, rd, int8ToUInt64(i8));
    }
    return advancePc(a, pc);
}


__device__ inline void executeLHU(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "lhu");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint64 rs1val = readX(a, rs1);
    uint16 u16 = readUInt16(a, uint64AddInt32(rs1val, imm));
    if (rd != 0) {
        writeX(a, rd, u16);
    }
    return advancePc(a, pc);
}


__device__ inline void executeLH(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "lh");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint64 rs1val = readX(a, rs1);
    int16 i16 = (int16)(readUInt16(a, uint64AddInt32(rs1val, imm)));
    if (rd != 0) {
        writeX(a, rd, int16ToUInt64((int64)(i16)));
    }
    return advancePc(a, pc);
}


__device__ inline void executeLW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "lw");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint64 rs1val = readX(a, rs1);
    int32 i32 = (int32)(readUInt32(a, uint64AddInt32(rs1val, imm)));
    if (rd != 0) {
        writeX(a, rd, int32ToUInt64(i32));
    }
    return advancePc(a, pc);
}


__device__ inline void executeLBU(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "lbu");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint64 rs1val = readX(a, rs1);
    uint8 u8 = readUInt8(a, uint64AddInt32(rs1val, imm));
    if (rd != 0) {
        writeX(a, rd, u8);
    }
    return advancePc(a, pc);
}


__device__ inline void executeSB(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sb");
    int32 imm = operandSimm12(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    uint64 rs1val = readX(a, rs1);
    uint64 rs2val = readX(a, rs2);
    writeUInt8(a, uint64AddInt32(rs1val, imm), (uint8)(rs2val));
    return advancePc(a, pc);
}


__device__ inline void executeSH(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sh");
    int32 imm = operandSimm12(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    uint64 rs1val = readX(a, rs1);
    uint64 rs2val = readX(a, rs2);
    writeUInt16(a, uint64AddInt32(rs1val, imm), (uint16)(rs2val));
    return advancePc(a, pc);
}


__device__ inline void executeSW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sw");
    int32 imm = operandSimm12(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    uint64 rs1val = readX(a, rs1);
    uint32 rs2val = (uint32)(readX(a, rs2));
    writeUInt32(a, uint64AddInt32(rs1val, imm), rs2val);
    return advancePc(a, pc);
}


__device__ inline void executeADDI(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "addi");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        int64 val = int64AddInt64((int64)(rs1val), (int64)(imm));
        writeX(a, rd, (uint64)(val));
    }
    return advancePc(a, pc);
}


__device__ inline void executeADDIW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "addiw");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    int32 rs1val = uint64ToInt32(readX(a, rs1));
    if (rd != 0) {
        int32 val = int32AddInt32(rs1val, imm);
        writeX(a, rd, int32ToUInt64(val));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSLTI(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "slti");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        if ((int64)(rs1val) < imm) {
            writeX(a, rd, 1);
        } else {
            writeX(a, rd, 0);
        }
    }
    return advancePc(a, pc);
}


__device__ inline void executeSLTIU(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sltiu");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        if (rs1val < int32ToUInt64(imm)) {
            writeX(a, rd, 1);
        } else {
            writeX(a, rd, 0);
        }
    }
    return advancePc(a, pc);
}


__device__ inline void executeXORI(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "xori");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        writeX(a, rd, rs1val ^ int32ToUInt64(imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeORI(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "ori");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        writeX(a, rd, rs1val | int32ToUInt64(imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeANDI(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "andi");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        writeX(a, rd, rs1val & int32ToUInt64(imm));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSLLI(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "slli");
    int32 imm = operandShamt6(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        writeX(a, rd, uint64ShiftLeft(rs1val, (uint32)(imm)));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSLLIW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "slliw");
    int32 imm = operandShamt5(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint32 rs1val = (uint32)(readX(a, rs1));
    if (rd != 0) {
        writeX(a, rd, int32ToUInt64((int32)(uint32ShiftLeft(rs1val, (uint32)(imm)))));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSRLI(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "srli");
    int32 imm = operandShamt6(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        writeX(a, rd, uint64ShiftRight(rs1val, (uint32)(imm)));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSRLW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "srlw");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    uint32 rs1val = (uint32)(readX(a, rs1));
    uint32 rs2val = (uint32)(readX(a, rs2));
    int32 rdval = (int32)(uint32ShiftRight(rs1val, rs2val));
    if (rd != 0) {
        writeX(a, rd, int32ToUInt64(rdval));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSRLIW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "srliw");
    int32 imm = operandShamt5(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint32 rs1val = (uint32)(readX(a, rs1));
    int32 rdval = (int32)(uint32ShiftRight(rs1val, (uint32)(imm)));
    if (rd != 0) {
        writeX(a, rd, int32ToUInt64(rdval));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSRAI(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "srai");
    int32 imm = operandShamt6(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        writeX(a, rd, (uint64)(int64ShiftRight((int64)(rs1val), (uint32)(imm))));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSRAIW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sraiw");
    int32 imm = operandShamt5(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    int32 rs1val = uint64ToInt32(readX(a, rs1));
    if (rd != 0) {
        writeX(a, rd, int32ToUInt64(int32ShiftRight(rs1val, (uint32)(imm))));
    }
    return advancePc(a, pc);
}


__device__ inline void executeADD(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "add");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        uint64 rs2val = readX(a, rs2);
        writeX(a, rd, uint64AddUInt64(rs1val, rs2val));
    }
    return advancePc(a, pc);
}


__device__ inline void executeADDW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "addw");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    int32 rs1val = uint64ToInt32(readX(a, rs1));
    int32 rs2val = uint64ToInt32(readX(a, rs2));
    if (rd != 0) {
        int32 val = int32AddInt32(rs1val, rs2val);
        writeX(a, rd, int32ToUInt64(val));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSUB(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sub");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        uint64 rs2val = readX(a, rs2);
        writeX(a, rd, uint64SubUInt64((int64)(rs1val), (int64) rs2val));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSUBW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "subw");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    int32 rs1val = uint64ToInt32(readX(a, rs1));
    int32 rs2val = uint64ToInt32(readX(a, rs2));
    if (rd != 0) {
        int32 val = int32SubInt32(rs1val, rs2val);
        writeX(a, rd, int32ToUInt64(val));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSLL(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sll");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        uint32 rs2val = (uint32)(readX(a, rs2));
        writeX(a, rd, uint64ShiftLeft(rs1val, rs2val));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSLLW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sllw");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    uint32 rs1val = (uint32)(readX(a, rs1));
    uint32 rs2val = (uint32)(readX(a, rs2));
    int32 rdval = (int32)(uint32ShiftLeft((uint32)(rs1val), rs2val));
    if (rd != 0) {
        writeX(a, rd, int32ToUInt64(rdval));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSLT(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "slt");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    if (rd != 0) {
        int64 rs1val = (int64)(readX(a, rs1));
        int64 rs2val = (int64)(readX(a, rs2));
        uint64 rdval = 0;
        if (rs1val < rs2val) {
            rdval = 1;
        }
        writeX(a, rd, rdval);
    }
    return advancePc(a, pc);
}


__device__ inline void executeSLTU(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sltu");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        uint64 rs2val = readX(a, rs2);
        uint64 rdval = 0;
        if (rs1val < rs2val) {
            rdval = 1;
        }
        writeX(a, rd, rdval);
    }
    return advancePc(a, pc);
}


__device__ inline void executeXOR(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "xor");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        uint64 rs2val = readX(a, rs2);
        writeX(a, rd, rs1val ^ rs2val);
    }
    return advancePc(a, pc);
}


__device__ inline void executeSRL(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "srl");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        uint64 rs2val = readX(a, rs2);
        writeX(a, rd, uint64ShiftRight(rs1val, (uint32)(rs2val)));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSRA(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sra");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    if (rd != 0) {
        int64 rs1val = (int64)(readX(a, rs1));
        uint32 rs2val = (uint32)(readX(a, rs2));
        writeX(a, rd, (uint64)(int64ShiftRight(rs1val, rs2val)));
    }
    return advancePc(a, pc);
}


__device__ inline void executeSRAW(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sraw");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    int32 rs1val = uint64ToInt32(readX(a, rs1));
    uint32 rs2val = (uint32)(readX(a, rs2));
    int32 rdval = int32ShiftRight(rs1val, rs2val);
    if (rd != 0) {
        writeX(a, rd, int32ToUInt64(rdval));
    }
    return advancePc(a, pc);
}


__device__ inline void executeOR(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "or");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        uint64 rs2val = readX(a, rs2);
        writeX(a, rd, rs1val | rs2val);
    }
    return advancePc(a, pc);
}


__device__ inline void executeAND(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "and");
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    if (rd != 0) {
        uint64 rs1val = readX(a, rs1);
        uint64 rs2val = readX(a, rs2);
        writeX(a, rd, rs1val & rs2val);
    }
    return advancePc(a, pc);
}


__device__ inline void executeFENCE(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "fence");
    return advancePc(a, pc);
}


__device__ inline void executeLWU(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "lwu");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint64 rs1val = readX(a, rs1);
    uint32 u32 = readUInt32(a, uint64AddInt32(rs1val, imm));
    if (rd != 0) {
        writeX(a, rd, u32);
    }
    return advancePc(a, pc);
}


__device__ inline void executeLD(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "ld");
    int32 imm = operandImm12(insn);
    uint8 rd = operandRd(insn);
    uint8 rs1 = operandRs1(insn);
    uint64 rs1val = readX(a, rs1);
    uint64 u64 = readUInt64(a, uint64AddInt32(rs1val, imm));
    if (rd != 0) {
        writeX(a, rd, u64);
    }
    return advancePc(a, pc);
}


__device__ inline void executeSD(UarchState *a, uint32 insn, uint64 pc) {
    dumpInsn(a, pc, insn, "sd");
    int32 imm = operandSimm12(insn);
    uint8 rs1 = operandRs1(insn);
    uint8 rs2 = operandRs2(insn);
    uint64 rs1val = readX(a, rs1);
    uint64 rs2val = readX(a, rs2);
    writeUInt64(a, uint64AddInt32(rs1val, imm), rs2val);
    return advancePc(a, pc);
}

/// \brief Returns true if the opcode field of an instruction matches the provided argument
__device__ inline bool insnMatchOpcode(uint32 insn, uint32 opcode) {
    return ((insn & 0x7f)) == opcode;
}

/// \brief Returns true if the opcode and funct3 fields of an instruction match the provided arguments
__device__ inline bool insnMatchOpcodeFunct3(uint32 insn, uint32 opcode, uint32 funct3) {
    const uint32 mask = (7 << 12) | 0x7f;
    return (insn & mask) == (uint32ShiftLeft(funct3, 12) | opcode);
}

/// \brief Returns true if the opcode, funct3 and funct7 fields of an instruction match the provided arguments
__device__ inline bool insnMatchOpcodeFunct3Funct7(uint32 insn, uint32 opcode, uint32 funct3, uint32 funct7) {
    const uint32 mask = (0x7f << 25) | (7 << 12) | 0x7f;
    return ((insn & mask)) == (uint32ShiftLeft(funct7, 25) | uint32ShiftLeft(funct3, 12) | opcode);
}

/// \brief Returns true if the opcode, funct3 and 6 most significant bits of funct7 fields of an instruction match the
/// provided arguments
__device__ inline bool insnMatchOpcodeFunct3Funct7Sr1(uint32 insn, uint32 opcode, uint32 funct3, uint32 funct7Sr1) {
    const uint32 mask = (0x3f << 26) | (7 << 12) | 0x7f;
    return ((insn & mask)) == (uint32ShiftLeft(funct7Sr1, 26) | uint32ShiftLeft(funct3, 12) | opcode);
}

// Decode and execute one instruction

__device__ inline void executeInsn(UarchState *a, uint32 insn, uint64 pc) {
    if (insnMatchOpcodeFunct3(insn, 0x13, 0x0)) {
        return executeADDI(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x3, 0x3)) {
        return executeLD(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x63, 0x6)) {
        return executeBLTU(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x63, 0x0)) {
        return executeBEQ(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x13, 0x7)) {
        return executeANDI(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x33, 0x0, 0x0)) {
        return executeADD(a, insn, pc);
    } else if (insnMatchOpcode(insn, 0x6f)) {
        return executeJAL(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7Sr1(insn, 0x13, 0x1, 0x0)) {
        return executeSLLI(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x33, 0x7, 0x0)) {
        return executeAND(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x23, 0x3)) {
        return executeSD(a, insn, pc);
    } else if (insnMatchOpcode(insn, 0x37)) {
        return executeLUI(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x67, 0x0)) {
        return executeJALR(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x1b, 0x0)) {
        return executeADDIW(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7Sr1(insn, 0x13, 0x5, 0x0)) {
        return executeSRLI(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x1b, 0x5, 0x0)) {
        return executeSRLIW(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x63, 0x1)) {
        return executeBNE(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x3, 0x2)) {
        return executeLW(a, insn, pc);
    } else if (insnMatchOpcode(insn, 0x17)) {
        return executeAUIPC(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x63, 0x7)) {
        return executeBGEU(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x3b, 0x0, 0x0)) {
        return executeADDW(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7Sr1(insn, 0x13, 0x5, 0x10)) {
        return executeSRAI(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x33, 0x6, 0x0)) {
        return executeOR(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x1b, 0x5, 0x20)) {
        return executeSRAIW(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x63, 0x5)) {
        return executeBGE(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x33, 0x0, 0x20)) {
        return executeSUB(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x3, 0x4)) {
        return executeLBU(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x1b, 0x1, 0x0)) {
        return executeSLLIW(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x33, 0x5, 0x0)) {
        return executeSRL(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x33, 0x4, 0x0)) {
        return executeXOR(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x23, 0x2)) {
       return executeSW(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x33, 0x1, 0x0)) {
        return executeSLL(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x63, 0x4)) {
        return executeBLT(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x23, 0x0)) {
        return executeSB(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x3b, 0x0, 0x20)) {
        return executeSUBW(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x13, 0x4)) {
        return executeXORI(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x33, 0x5, 0x20)) {
        return executeSRA(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x3, 0x5)) {
        return executeLHU(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x23, 0x1)) {
        return executeSH(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x3b, 0x5, 0x0)) {
        return executeSRLW(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x3, 0x6)) {
        return executeLWU(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x3b, 0x1, 0x0)) {
        return executeSLLW(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x3, 0x0)) {
        return executeLB(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x33, 0x3, 0x0)) {
        return executeSLTU(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x3b, 0x5, 0x20)) {
        return executeSRAW(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x3, 0x1)) {
        return executeLH(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x13, 0x6)) {
        return executeORI(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x13, 0x3)) {
        return executeSLTIU(a, insn, pc);
    } else if (insnMatchOpcodeFunct3Funct7(insn, 0x33, 0x2, 0x0)) {
        return executeSLT(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0x13, 0x2)) {
        return executeSLTI(a, insn, pc);
    } else if (insnMatchOpcodeFunct3(insn, 0xf, 0x0)) {
        return executeFENCE(a, insn, pc);
    }
    a->trap = 253;    
//    throw std::runtime_error("illegal instruction");
}

__device__ UArchStepStatus uarch_step(UarchState *a) {
    // This must be the first read in order to match the first log access in machine::verify_state_transition
    uint64 cycle = readCycle(a);
    // do not advance if cycle will overflow
    if (cycle == UINT64_MAX) {
        return CycleOverflow;
    }
    // do not advance if machine is halted
    if (readHaltFlag(a)) {
        return UArchHalted;
    }
    // execute next instruction
    uint64 pc = readPc(a);
    uint32 insn = readUInt32(a, pc);
    executeInsn(a, insn, pc);
    cycle = cycle + 1;
    writeCycle(a, cycle);
    return Success;
}


struct Input {
    uint64 access_paddr[16];
    uint64 access_val[16];
    uint8 access_readWriteEnd[16];
};

typedef struct Input Input;

__global__ void rv64i() {
   struct Input input;
   UarchState state;
   state.access_pointer = 0;
   state.trap = 0;
   for (int i = 0; i < 16; i++) {
      state.access_paddr[i] = input.access_paddr[i];
      state.access_val[i] = input.access_val[i];
      state.access_readWriteEnd[i] = input.access_readWriteEnd[i];
   }
   enum UArchStepStatus ret = uarch_step(&state);
   int retval = 0;
   if (state.access_readWriteEnd[state.access_pointer] != 2) {
     retval = 22;
   } else if (state.trap > 0) {
     retval = state.trap;
   } else if (ret != Success) {
     retval = 1;
   } else if (state.access_pointer > 16) {
     retval = 16;
   } else if (state.access_readWriteEnd[state.access_pointer] != 2) {
     retval = 17;
   } else {
     retval = 0;
   }
   return;
}

int main() {
    rv64i<<<5,1>>>();
    return 0;
}

